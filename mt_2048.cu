#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h" 
#include <iostream>
#include "gputimer.h"
#include <stdio.h>

//#include <time>

using namespace std;
__device__ int combine(int * l_state, int turn){
    int score=0;
    bool if_move=false; 

    if (turn==0){ //left
        for(int i=0;i<4;i++){
            for(int t=0;t<3;t++){
                int j =0, k=0;
                if(t!=1){
                    for(j=0;j<4;j++){
                        if(l_state[4*i+j]!=0){
                            l_state[4*i+k]=l_state[4*i+j];
                            k++;
                        }
                    }
                    while(k<4){l_state[4*i+k]=0; k++; if_move = true;}
                }
                else
                    for(j=0;j<3;j++){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j+1]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*i+j+1]=0;
                            j++;
                        }
                    }
            }
        }
    }
    if (turn==1){  //down
        for(int j=0;j<4;j++){
            for(int t=0;t<3;t++){
                int i =3, k=3;
                if(t!=1){
                    for(i=3;i>=0;i--){
                        if(l_state[4*i+j]!=0){
                            l_state[4*k+j]=l_state[4*i+j];
                            k--;

                        }
                    }
                    while(k>=0){l_state[4*k+j]=0; k--; if_move = true;}
                }
                else
                    for(i=3;i>=0;i--){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i-1)+j]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*(i-1)+j]=0;
                            i--;
                        }
                    }
            }
        }
    }
    if (turn==2){   //right
        for(int i=0;i<4;i++){
            for(int t=0;t<3;t++){
                int j =3, k=3;
                if(t!=1){
                    for(j=3;j>=0;j--){
                        if(l_state[4*i+j]!=0){
                            l_state[4*i+k]=l_state[4*i+j];
                            k--;
                        }
                    }
                    while(k>=0){l_state[4*i+k]=0; k--; if_move = true;}
                }
                else
                    for(j=3;j>0;j--)
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j-1]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*i+j-1]=0;
                            j--;
                        }
            }
        }
    }
    if (turn==3){//up
        for(int j=0;j<4;j++){
            for(int t=0;t<3;t++){
                int i =0, k=0;
                if(t!=1){
                    for(i=0;i<4;i++){
                        if(l_state[4*i+j]!=0){
                            l_state[4*k+j]=l_state[4*i+j];
                            k++;
                        }
                    }
                    while(k<4){l_state[4*k+j]=0; k++; if_move = true;}
                }
                else
                    for(i=0;i<3;i++){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i+1)+j]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*(i+1)+j]=0;
                            i++;
                        }
                    }
            }
        }
    }
    return score;
}


// over:0
__device__ int judge_over(const int *state)
{
    //bool alive=false;
    int zero_num=0;
    bool can_combine = false;
    for(int i=0;i<4;i++){
        for(int j=0;j<4;j++){
            if(state[4*i+j]==0) zero_num+=1;
            if(i<3 && j<3 && (state[4*i+j]==state[4*i+j+1] || state[4*i+j]==state[4*(i+1)+j]))
                can_combine=true;
        }
    }
    if(zero_num==0){
        if(!can_combine) return -1;
        else return zero_num;
    }            
    return true;
}

__device__ hiprandState newBlock(int * state, hiprandState curand_state, int No)
{

    int new_num=1;
    int r_num = hiprand(&curand_state)%10;
    if(r_num>8) new_num=2;
    bool ok=false;
    while(!ok){
        int r_place = hiprand(&curand_state)%16;
        //printf("No is %d, r_num is %d, r_place is %d \n", No, r_num, r_place);
        if(state[r_place]==0){
            state[r_place]=new_num;
            ok=true;
        }
    } 
    return curand_state;
}

__device__ void print_array(int *array, int size){
    for(int i=0; i<size; i++)
        printf("%d, ",array[i]);
    printf("\n");
}

__global__ void run_2048(const int * d_state, int *d_result, int search_depth, long seed ){
    int No = blockIdx.x * blockDim.x + threadIdx.x; 

    hiprandState curand_state;
    hiprand_init(seed-No, 0 ,0, &curand_state);

    int tid  = threadIdx.x;

    int init_state = threadIdx.x;
    int depth = search_depth;

    int  l_state[16];
    for(int i=0;i<16;i++)
        l_state[i] = d_state[i];
    int result=1;
    int score=0;
    int turn =0;
    int zero_num=16;

    while(depth && zero_num!=-1)
    {           
        zero_num = judge_over(l_state);
        if(zero_num>0)
            curand_state=newBlock(l_state, curand_state,No);
        turn = init_state%4;
        init_state/=4;
        depth-=1;
        result=combine(l_state, turn);
        score+=result;
    }

    int count=0;
    while(zero_num!=-1){
        count+=1;
        zero_num= judge_over(l_state);
        if(zero_num>0)
            curand_state=newBlock(l_state, curand_state,No);
        turn = hiprand(&curand_state)%4;
        result=combine(l_state, turn);
        score+=result;
    }
    atomicAdd(&(d_result[tid]),score);
    
 /*   printf("score3 is %d \n",score);
    printf("zero_num : %d \n",zero_num);
    printf("Block_id:%d, threadIdx.x: %d, atomicAdd:%d \n", 
                blockIdx.x, threadIdx.x, d_result[threadIdx.x]);
    print_array(l_state,16);*/
    
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init ( seed, id , 0, &state[id] );
}


void print_array2(int *array, int size){
    for(int i=0; i<size; i++)
        printf("%d, ",array[i]);
    printf("\n");
}

int get_best_turn(int *h_state, int exp_num=1000, int search_depth=2){
    int *d_state;
    //int h_state[16]={0};

    const int ARRAY_BYTES = 16*sizeof(int);
    hipMalloc((void **) &d_state, ARRAY_BYTES);
    hipMemset((void **) d_state, 0, ARRAY_BYTES);
    hipMemcpy(d_state, h_state, ARRAY_BYTES, hipMemcpyHostToDevice);

    long clock_for_rand = clock();

    const int search_kinds = (1<<(search_depth*2));

    int *d_result;
    int *h_result=new int[search_kinds];
    const int RESULT_BYTES = search_kinds*sizeof(int);
    for(int i=0; i<search_kinds; i++) h_result[i]=0;

    hipMalloc((void **) &d_result, RESULT_BYTES); 
    hipMemset((void **) d_result, 0, RESULT_BYTES);
    hipMemcpy(d_result, h_result, RESULT_BYTES , hipMemcpyHostToDevice);

    //timer.Start();
    run_2048<<<exp_num, search_kinds >>>(d_state, d_result, search_depth, clock_for_rand);
    //timer.Stop();

    hipMemcpy(h_result, d_result, RESULT_BYTES, hipMemcpyDeviceToHost);

    int max=0;
    int best_way=0;
    for(int i=0;i<search_kinds;i++){
        if(h_result[i]>max){
            max = h_result[i];
            best_way = i;
        }
    }

    int best_turn = best_way%4;

    print_array2(h_result, search_kinds);

    printf("Best way is %d , Best score = %d , Best turn is %d \n", 
                best_way, h_result[best_way], best_turn);

    hipFree(d_state);
    hipFree(d_result);
    return best_turn;
}


int main(int argc,char *argv[]){
 //   printf("Total amount of global memory: %d bytes",deviceProp.total)


    GpuTimer timer;

    int *d_state;
    int h_state[16]={0};

    const int ARRAY_BYTES = 16*sizeof(int);
    hipMalloc((void **) &d_state, ARRAY_BYTES);
    hipMemset((void **) d_state, 0, ARRAY_BYTES);

    long clock_for_rand = clock();
    cout<<"get arg num is "<<argc<<endl;

    int i=0, k=0;
    while(argc>1 && argv[1][i]!='\0')
    {
        if(argv[1][i]!=',')
            h_state[k]=h_state[k]*10+argv[1][i]-'0';
        else{
            k+=1;
            if(k>=16)
                cout<<"num of number exceed 16!"<<endl;
        }
        i=i+1;
    }
    cout<<"input h_state is:";
    print_array2(h_state,16);


    timer.Start();
    get_best_turn(h_state,10000,2);
    timer.Stop();

    //hipMemcpy(h_state, d_state, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("Time elapsed = %g ms\n", timer.Elapsed());
    return 0;
}