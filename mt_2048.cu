#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h" 
#include <iostream>
#include "gputimer.h"

using namespace std;
__device__ int combine(int * l_state, int turn){
	int score=0;
	if (turn==0){
		for(int i=0;i<4;i++){
			for(int t=0;t<3;t++){
				int j =0, k=0;
				if(t!=1){
					for(j=0;j<4;j++){
						if(l_state[4*i+j]!=0){
							l_state[4*i+k]=l_state[4*i+j];
							k++;
						}
					}
					while(k<4){l_state[4*i+k]=0; k++; }
				}
				else
					for(j=0;j<3;j++){
						if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j+1]){
							l_state[4*i+j]+=1;
							score+=1;
							l_state[4*i+j+1]=0;
							j++;
						}
					}
			}
		}
	}
	if (turn==1){  //right
		for(int j=0;j<4;j++){
			for(int t=0;t<3;t++){
				int i =3, k=3;
				if(t!=1){
					for(i=3;i>=0;i--){
						if(l_state[4*i+j]!=0){
							l_state[4*k+j]=l_state[4*i+j];
							k--;
						}
					}
					while(k>=0){l_state[4*k+j]=0; k--; }
				}
				else
					for(i=3;i>=0;i--){
						if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i-1)+j]){
							l_state[4*i+j]+=1;
							score+=1;
							l_state[4*(i-1)+j]=0;
							i--;
						}
					}
			}
		}
	}
	if (turn==2){   //down
		for(int i=0;i<4;i++){
			for(int t=0;t<3;t++){
				int j =3, k=3;
				if(t!=1){
					for(j=3;j>=0;j--){
						if(l_state[4*i+j]!=0){
							l_state[4*i+k]=l_state[4*i+j];
							k--;
						}
					}
					while(k>=0){l_state[4*i+k]=0; k--; }
				}
				else
					for(j=3;j>0;j--)
						if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j-1]){
							l_state[4*i+j]+=1;
							score+=1;
							l_state[4*i+j-1]=0;
							j--;
						}
			}
		}
	}
	if (turn==3){//left
		for(int j=0;j<4;j++){
			for(int t=0;t<3;t++){
				int i =0, k=0;
				if(t!=1){
					for(i=0;i<4;i++){
						if(l_state[4*i+j]!=0){
							l_state[4*k+j]=l_state[4*i+j];
							k++;
						}
					}
					while(k<4){l_state[4*k+j]=0; k++; }
				}
				else
					for(i=0;i<3;i++){
						if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i+1)+j]){
							l_state[4*i+j]+=1;
							score+=1;
							l_state[4*(i+1)+j]=0;
							i++;
						}
					}
			}
		}
	}
	return score;
}

__device__ bool judge_over(int *state)
{
	//bool alive=false;
	for(int i=0;i<3;i++){
		for(int j=0;j<3;j++){
			if(state[4*i+j]==0) return false;
			if(state[4*i+j]==state[4*i+j+1] || state[4*i+j]==state[4*(i+1)+j])
				return false;
		}	
	}
	return true;
}

__device__ hiprandState newBlock(int * state, hiprandState curand_state)
{

	int new_num=1;
	int r_num = hiprand(&curand_state)%10;
	if(r_num>8) new_num=2;
	bool ok=false;
	while(!ok){
		int r_place = hiprand(&curand_state)%16;
		if(state[r_place]==0){
			state[r_place]=new_num;
			ok=true;
		}
	} 
	return curand_state;
}

__device__ void print_array(int *array, int size){
	for(int i=0; i<size; i++)
		printf("%d, ",array[i]);
	printf("\n");
}

__global__ void run_2048(int * d_state, int *d_result, int search_depth, long seed ){
	int No = blockIdx.x * blockDim.x + threadIdx.x; 

	hiprandState curand_state;
	hiprand_init((seed-No), 0 ,0,&curand_state);
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int init_state = threadIdx.x;
	int depth = search_depth;

	int  l_state[16];
	for(int i=0;i<16;i++)
		l_state[i] = d_state[i];
	int result=1;
	int score=0;
	int turn =0;
	bool over_flag=false;

	while(depth && !over_flag)
	{ 	
		curand_state=newBlock(l_state, curand_state);
		turn = init_state%4;
		init_state/=4;
		depth-=1;
		result=combine(l_state, turn);
		over_flag = judge_over(l_state);
		score+=result;
	}

	if(No==0){
		printf("score is %d \n",score);
		printf("over_flag is %d \n",over_flag);
		print_array(l_state,16);
	}
	__syncthreads();

	while(!over_flag){
		if(No==0){
			printf("score is %d \n",score);
			printf("over_flag : %d \n",over_flag);
			print_array(l_state,16);
		}
		curand_state=newBlock(l_state, curand_state);
		turn = hiprand(&curand_state)%4;
		result=combine(l_state, turn);
		over_flag= judge_over(l_state);
		score+=result;
	}

	if(No==0){
		printf("score is %d \n",score);
		printf("over_flag : %d \n",over_flag);
		print_array(l_state,16);
	}
	atomicAdd(&d_result[blockIdx.x],score);
	__syncthreads();

}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init ( seed, id , 0, &state[id] );
}


__host__ void print_array2(int *array, int size){
	for(int i=0; i<size; i++)
		printf("%d, ",array[i]);
	printf("\n");
}

int main(int argc,char *argv[]){
	GpuTimer timer;

	int *d_state;
	int h_state[16]={0};

	const int ARRAY_BYTES = 16*sizeof(int);
	hipMalloc((void **) &d_state, ARRAY_BYTES);
	hipMemset((void **) d_state, 0, ARRAY_BYTES);

	long clock_for_rand = clock();
	cout<<"get arg num is "<<argc<<endl;
	int i=0, k=0;

	while(argc>1 && argv[1][i]!=0)
	{
		if(argv[1][i]!=',')
			h_state[k]=h_state[k]*10+argv[1][i]-'0';
		else{
			k+=1;
			if(k>=16)
				cout<<"num of number exceed 16!"<<endl;
		}
	}
	hipMemcpy(h_state, d_state, ARRAY_BYTES, hipMemcpyHostToDevice);

	int exp_num=1000;
	int search_depth=3;
	int search_kinds = (1<<search_depth);

	int *d_result;
	int *h_result = new int[search_kinds];
	for(int i=0;i<search_kinds;i++) h_result[i]=0;
	hipMalloc((void **) &d_result, search_kinds*sizeof(int)); 
	hipMemset((void **) d_state, 0, search_kinds*sizeof(int));

	hipMemcpy(h_result, d_result, ARRAY_BYTES, hipMemcpyHostToDevice);


	timer.Start();
	run_2048<<<exp_num, search_kinds >>>(d_state, d_result, search_depth, clock_for_rand);
	timer.Stop();

	hipMemcpy(h_state, d_state, ARRAY_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(h_result, d_result, ARRAY_BYTES, hipMemcpyDeviceToHost);

	int max=0;
	int best_way=0;
	for(int i=0;i<search_kinds;i++){
		if(h_result[i]>max){
			max = h_result[i];
			best_way = i;
		}
	}

	int best_turn = best_way%4;

	//print_array2(h_state, 16);

	printf("Time elapsed = %g ms\n", timer.Elapsed());
	printf("Best way is %d , Best score = %d , Best turn is %d \n", 
				best_way, h_result[best_way], best_turn);
	hipFree(d_state);
	return 0;
}