#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h" 
#include <iostream>
#include "gputimer.h"
#include <stdio.h>

//#include <time>

using namespace std;
__device__ int combine(int * l_state, int turn){
    int score=0;
    bool can_move=false; 
    bool if_move=false;
    if (turn==0){ //left
        for(int i=0;i<4;i++){
            for(int t=0;t<3;t++){
                int j =0, k=0; 
                can_move=false;
                if(t!=1){
                    for(j=0;j<4;j++){
                        if(l_state[4*i+j]!=0){
                            l_state[4*i+k]=l_state[4*i+j];
                            k++;
                            if(can_move) if_move=true;
                        }
                        else can_move=true;
                    }
                    while(k<4){l_state[4*i+k]=0; k++;}
                }
                else
                    for(j=0;j<3;j++){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j+1]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*i+j+1]=0;
                            j++;
                        }
                    }
            }
        }
    }
    if (turn==1){  //down
        for(int j=0;j<4;j++){
            for(int t=0;t<3;t++){
                int i =3, k=3;
                can_move=false;
                if(t!=1){
                    for(i=3;i>=0;i--){
                        if(l_state[4*i+j]!=0){
                            l_state[4*k+j]=l_state[4*i+j];
                            k--;
                            if(can_move) if_move=true;
                        }
                        else can_move=true;
                    }
                    while(k>=0){l_state[4*k+j]=0; k--;}
                }
                else
                    for(i=3;i>=0;i--){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i-1)+j]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*(i-1)+j]=0;
                            i--;
                        }
                    }
            }
        }
    }
    if (turn==2){   //right
        for(int i=0;i<4;i++){
            for(int t=0;t<3;t++){
                int j =3, k=3;
                can_move=false;
                if(t!=1){
                    for(j=3;j>=0;j--){
                        if(l_state[4*i+j]!=0){
                            l_state[4*i+k]=l_state[4*i+j];
                            k--;
                            if(can_move) if_move=true;
                        }
                        else can_move=true;
                    }
                    while(k>=0){l_state[4*i+k]=0; k--;}
                }
                else
                    for(j=3;j>0;j--)
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*i+j-1]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*i+j-1]=0;
                            j--;
                        }
            }
        }
    }
    if (turn==3){//up
        for(int j=0;j<4;j++){
            for(int t=0;t<3;t++){
                int i =0, k=0;
                can_move=false;
                if(t!=1){
                    for(i=0;i<4;i++){
                        if(l_state[4*i+j]!=0){
                            l_state[4*k+j]=l_state[4*i+j];
                            k++;
                            if(can_move) if_move=true;
                        }
                        else can_move=true;
                        
                    }
                    while(k<4){l_state[4*k+j]=0; k++;}
                }
                else
                    for(i=0;i<3;i++){
                        if(l_state[4*i+j] && l_state[4*i+j]==l_state[4*(i+1)+j]){
                            l_state[4*i+j]+=1;
                            score+=1;
                            l_state[4*(i+1)+j]=0;
                            i++;
                        }
                    }
            }
        }
    }
    if(if_move==false && score==0) score=-1;
    return score;
}


// over:0
__device__ int judge_over(const int *state)
{
    //bool alive=false;
    int zero_num=0;
    bool can_combine = false;
    for(int i=0;i<4;i++){
        for(int j=0;j<4;j++){
            if(state[4*i+j]==0) zero_num+=1;
            if((j<3 && state[4*i+j]==state[4*i+j+1]) ||
                (i<3 && state[4*i+j]==state[4*(i+1)+j]))
                can_combine=true;
        }
    }
    if(zero_num==0){
        if(!can_combine) return -1;
        else return zero_num;
    }            
    return zero_num;
}

__device__ hiprandState newBlock(int * state, hiprandState curand_state, int No)
{

    int new_num=1;
    int r_num = hiprand(&curand_state)%10;
    if(r_num>8) new_num=2;
    bool ok=false;
    while(!ok){
        int r_place = hiprand(&curand_state)%16;
        //printf("No is %d, r_num is %d, r_place is %d \n", No, r_num, r_place);
        if(state[r_place]==0){
            state[r_place]=new_num;
            ok=true;
        }
    } 
    return curand_state;
}

__device__ void print_array(int *array, int size){
    for(int i=0; i<size; i++)
        printf("%d, ",array[i]);
    printf("\n");
}

__device__ void print_array_2d(int *array, int x_size, int y_size){
    for(int i=0; i<x_size; i++){
        for(int j=0; j<y_size; j++)
            printf("%d, ",array[4*i+j]);
        printf("\n");
    }
    //printf("\n");
}
__global__ void run_2048(const int * d_state, int *d_result, int search_depth, long seed ){
    int No = blockIdx.x * blockDim.x + threadIdx.x; 

    hiprandState curand_state;
    hiprand_init(seed-No, 0 ,0, &curand_state);

    int tid  = threadIdx.x;

    int init_state = threadIdx.x;
    int depth = search_depth;

    int  l_state[16];
    for(int i=0;i<16;i++)
        l_state[i] = d_state[i];
    int result=1;
    int score=0;
    int turn =0;
    int zero_num=16;

    zero_num = judge_over(l_state);
    while(depth && zero_num!=-1)
    {           
        //if(No==0){printf("After new: turn:%d  \n",turn); print_array_2d(l_state,4,4);}
        turn = init_state%4;
        init_state/=4;
        depth-=1;
        result=combine(l_state, turn);
        if(result==-1 && depth==search_depth-1) score-=100;
        zero_num = judge_over(l_state);
        score+=result*(result>0);
        if(zero_num>0 && result!=-1)
            curand_state=newBlock(l_state, curand_state,No);
        //if(No==0){printf("turn:%d score:%d \n",turn,score); print_array_2d(l_state,4,4);}
    }

    int count=0;
    while(zero_num!=-1){
        count+=1;
        //if(No==0){printf("After new: turn:%d  \n",turn); print_array_2d(l_state,4,4);}
        turn = hiprand(&curand_state)%4;
        result=combine(l_state, turn);
        score+=result*(result>0);
        //if(No==0){printf("turn:%d score:%d \n",turn,score); print_array_2d(l_state,4,4);}
        zero_num= judge_over(l_state);
        if(zero_num>0 && result!=-1)
            curand_state=newBlock(l_state, curand_state,No);
    }
    atomicAdd(&(d_result[tid]),score);
    
 /*   printf("score3 is %d \n",score);
    printf("zero_num : %d \n",zero_num);
    printf("Block_id:%d, threadIdx.x: %d, atomicAdd:%d \n", 
                blockIdx.x, threadIdx.x, d_result[threadIdx.x]);
    print_array(l_state,16);*/
    
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x  + blockIdx.x * blockDim.x;
    hiprand_init ( seed, id , 0, &state[id] );
}


void print_array2(int *array, int size){
    for(int i=0; i<size; i++)
        printf("%d, ",array[i]);
    printf("\n");
}

int get_best_turn(int *h_state, int exp_num=5000, int search_depth=2, bool print_flag=true){
    int *d_state;
    //int h_state[16]={0};

    const int ARRAY_BYTES = 16*sizeof(int);
    hipMalloc((void **) &d_state, ARRAY_BYTES);
    hipMemset((void **) d_state, 0, ARRAY_BYTES);
    hipMemcpy(d_state, h_state, ARRAY_BYTES, hipMemcpyHostToDevice);

    long clock_for_rand = clock();

    const int search_kinds = (1<<(search_depth*2));

    int *d_result;
    int *h_result=new int[search_kinds];
    const int RESULT_BYTES = search_kinds*sizeof(int);
    for(int i=0; i<search_kinds; i++) h_result[i]=0;

    hipMalloc((void **) &d_result, RESULT_BYTES); 
    hipMemset((void **) d_result, 0, RESULT_BYTES);
    hipMemcpy(d_result, h_result, RESULT_BYTES , hipMemcpyHostToDevice);

    //timer.Start();
    run_2048<<<exp_num, search_kinds >>>(d_state, d_result, search_depth, clock_for_rand);
    //timer.Stop();

    hipMemcpy(h_result, d_result, RESULT_BYTES, hipMemcpyDeviceToHost);

    int max=-10000000;
    int best_way=0;
    for(int i=0;i<search_kinds;i++){
        if(h_result[i]>max){
            max = h_result[i];
            best_way = i;
        }
    }

    int best_turn = best_way%4;

    if(print_flag){
        print_array2(h_result, search_kinds);
        printf("Best way is %d , Best score = %d , Best turn is %d \n", 
                best_way, h_result[best_way], best_turn);
    }

    hipFree(d_state);
    hipFree(d_result);
    return best_turn;
}


int main(int argc,char *argv[]){
 //   printf("Total amount of global memory: %d bytes",deviceProp.total)
    GpuTimer timer;

    int *d_state;
    int h_state[16]={0};

    const int ARRAY_BYTES = 16*sizeof(int);
    hipMalloc((void **) &d_state, ARRAY_BYTES);
    hipMemset((void **) d_state, 0, ARRAY_BYTES);

    long clock_for_rand = clock();
    //cout<<"get arg num is "<<argc<<endl;

    int i=0, k=0;
    while(argc>1 && argv[1][i]!='\0')
    {
        if(argv[1][i]!=',')
            h_state[k]=h_state[k]*10+argv[1][i]-'0';
        else{
            k+=1;
            if(k>=16)
                cout<<"num of number exceed 16!"<<endl;
        }
        i=i+1;
    }
    bool print_flag=true;
    if (argc>2 && argv[2][0]=='1') print_flag=false;

    int exp_num = 3000;
    int search_depth=2;
    if (argc>3){
        exp_num=0; int i=0;  
        while(argv[3][i]!='\0'){
            exp_num=exp_num*10+argv[3][i]-'0';
            i++;
        }
    }

    if(argc>4) search_depth = argv[4][0]-'0';
    timer.Start();
    int best_turn = get_best_turn(h_state, exp_num, search_depth, print_flag);
    timer.Stop();

    //hipMemcpy(h_state, d_state, ARRAY_BYTES, hipMemcpyDeviceToHost);
   if(print_flag){
        cout<<"input h_state is:";
        print_array2(h_state,16);
        printf("Time elapsed = %g ms\n", timer.Elapsed());
    }
    return best_turn;
}